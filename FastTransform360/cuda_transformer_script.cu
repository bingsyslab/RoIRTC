#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream> // temp

using namespace std;

texture<float, 2, hipReadModeElementType> tex_ref; // texure reference must be a global variable

__device__ float CUDAradians(float degrees) {return 0.01745329252 * degrees;}

__global__ void CUDARotateKernel(void) {}

__global__ void CUDATransformKernel(float* cuda_transformed_data,
                                    int width,
                                    int height,
                                    float y,
                                    float p,
                                    float r,
                                    float ratio,
                                    int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    // float u = (index % width) / __int2float_rn(width);
    float u = __fdividef(__int2float_rn(index % width), __int2float_rn(width));
    // float v = (index / width) / __int2float_rn(height);
    float v = __fdividef(__int2float_rn(index / width), __int2float_rn(height));
    // u = 0.75 * u - 0.375;
    u = __fmaf_rn(0.75, u, -0.375);
    // Try not to use if-else statement in kernel in future
    // Like use two kernels to calculate up-half and down-half seperately
    // And use fixed 'ratio' value to make calcualtion faster
    if (v < ratio) {
      // v = 0.25 - v / (2.0 * ratio);
      v = 0.25 - __fdividef(v, 2.0 * ratio);
    } else {
      // v = ((1.0 + ratio) * 0.5 - v) / (2.0 * (1.0 - ratio));
      v = __fdividef(__fmaf_rn(1.0 + ratio, 0.5, -v), 2.0 * (1.0 - ratio));
    }
    // u, v, 0 => su, cu, sv, cv, sin(0), cos(0)
    float su, cu, sv, cv;
    sincospif(2.0 * u, &su, &cu);
    sincospif(v, &sv, &cv);
    /* mat3(cu,    0,    -su,
            0,     1,     0,
            su,    0,     cu)
       mat3(1,     0,     0,
            0,     cv,    sv,
            0,    -sv,    cv) */
    /* float d00 = cu,      d01 = 0,   d02 = -su,
             d10 = su * sv, d11 = cv,  d12 = cu * sv,
             d20 = su * cv, d21 = -sv, d22 = cu * cv;
    /* mat3(cz,    sz,    0,
           -sz,    cz,    0,
            0,     0,     1) */
    // rotation matrix calcualtion
    /* float p00 = cu * cz + su * sv * sz,
             p01 = cv * sz,
             p02 = -su * cz + cu * sv * sz,
             p10 = cu * -sz + su * sv * cz,
             p11 = cv * cz,
             p12 = -su * -sz + cu * sv * cz,
             p20 = su * cv,
             p21 = -sv,
             p22 = cu * cv; */
    // vector 'ps'
    // float v00 = su * cv, v01 = -sv, v02 = cu * cv;
    // 0, 0, 1 -> 0, 0, -1
    // float v00 = -su * cv, v01 = sv, v02 = -cu * cv;
    float sy, cy, sp, cp, sr, cr;
    if (index < ratio * n) {
      // rotationMatrix(radians(vec3(pitch, yaw, -roll))) * ps
      y = CUDAradians(y);
      p = CUDAradians(p);
      r = CUDAradians(-r);
      sincosf(y, &sy, &cy);
      sincosf(p, &sp, &cp);
      sincosf(r, &sr, &cr);
      // rotation matrix calcualtion
      /* float p00 = cy * cr + sy * sp * sr,
               p01 = cp * sr,
               p02 = -sy * cr + cy * sp * sr,
               p10 = cy * -sr + sy * sp * cr,
               p11 = cp * cr,
               p12 = -sy * -sr + cy * sp * cr,
               p20 = sy * cp,
               p21 = -sp,
               p22 = cy * cp; */
      // vector 'ps'
      // float v00 = su * cv, v01 = -sv, v02 = cu * cv;
      // 0, 0, 1 -> 0, 0, -1
      // float v00 = -su * cv, v01 = sv, v02 = -cu * cv;
      float vx = (cy * cr + sy * sp * sr) * -su * cv + (cy * -sr + sy * sp * cr) * sv + sy * cp * -cu * cv,
            vy = cp * sr * -su * cv + cp * cr * sv + -sp * -cu * cv,
            vz = (-sy * cr + cy * sp * sr) * -su * cv + (-sy * -sr + cy * sp * cr) * sv + cy * cp * -cu * cv;
      u = atan2f(vx, vz);
      if (u < 0.0) u = u + M_PI + M_PI;
      v = acosf(vy);
      u = __fdividef(u, M_PI + M_PI);
      v = __fdividef(v, M_PI);
    } else {
      // rotationMatrix(radians(vec3(-pitch, yaw + 180.0, 90.0 + roll))) * ps
      y = CUDAradians(180.0 + y);
      p = CUDAradians(-p);
      r = CUDAradians(90.0 + r);
      sincosf(y, &sy, &cy);
      sincosf(p, &sp, &cp);
      sincosf(r, &sr, &cr);
      // rotation matrix calcualtion
      /* float p00 = cy * cr + sy * sp * sr,
               p01 = cp * sr,
               p02 = -sy * cr + cy * sp * sr,
               p10 = cy * -sr + sy * sp * cr,
               p11 = cp * cr,
               p12 = -sy * -sr + cy * sp * cr,
               p20 = sy * cp,
               p21 = -sp,
               p22 = cy * cp; */
      // vector 'ps'
      // float v00 = su * cv, v01 = -sv, v02 = cu * cv;
      // 0, 0, 1 -> 0, 0, -1
      // float v00 = -su * cv, v01 = sv, v02 = -cu * cv;
      float vx = (cy * cr + sy * sp * sr) * -su * cv + (cy * -sr + sy * sp * cr) * sv + sy * cp * -cu * cv,
            vy = cp * sr * -su * cv + cp * cr * sv + -sp * -cu * cv,
            vz = (-sy * cr + cy * sp * sr) * -su * cv + (-sy * -sr + cy * sp * cr) * sv + cy * cp * -cu * cv;
      u = atan2f(vx, vz);
      if (u < 0.0) u = u + M_PI + M_PI;
      v = acosf(vy);
      u = __fdividef(u, M_PI + M_PI);
      v = __fdividef(v, M_PI);
    }
    __syncthreads();
    cuda_transformed_data[index] = tex2D(tex_ref, u, v);
  }
}

__global__ void CUDAReformKernel(void) {}

int CUDARotateWrapper() {return 0;}

int CUDATransformWrapper(const uint8_t* data,
                         int width,
                         int height,
                         int target_width,
                         int target_height,
                         float yaw,
                         float pitch,
                         float roll,
                         uint8_t* transformed_data,
                         int flag) {
  // Use switch(flag) {} in future
  float ratio = target_width / (3.0 * target_height);
  int n = target_width * target_height;
  int tex_size = width * height;
  float* tex = (float*)malloc(tex_size * sizeof(float)); // temp
  for (int i = 0; i < tex_size; i++) *(tex + i) = (float)*(data + i); // temp
  hipArray* cuArray;
  hipMallocArray(&cuArray, &tex_ref.channelDesc, width, height);
  hipMemcpyToArray(cuArray,
                    0, 0,
                    tex,
                    tex_size * sizeof(float),
                    hipMemcpyHostToDevice);
  tex_ref.addressMode[0] = hipAddressModeWrap;
  tex_ref.addressMode[1] = hipAddressModeWrap;
  tex_ref.filterMode = hipFilterModeLinear;
  tex_ref.normalized = true;
  hipBindTextureToArray(tex_ref, cuArray);
  // Use float* to debug, future use int*
  float* cuda_transformed_data = NULL;
  if (hipMalloc((void**)&cuda_transformed_data,
                  n * sizeof(float)) != hipSuccess) {
    cerr << "hipMalloc() Failed: cuda_transformed_data" << endl;
    return 1;
  }
  // To be modified depends on memory usage of each thread
  int n_threads = min(256, target_height);
  int n_blocks = (n + n_threads - 1) / n_threads;
  CUDATransformKernel<<<n_blocks, n_threads>>>(cuda_transformed_data,
                                               target_width,
                                               target_height,
                                               yaw,
                                               pitch,
                                               roll,
                                               ratio, n);
  if (hipDeviceSynchronize() != hipSuccess) {
    cerr << "hipDeviceSynchronize() Failed" << endl;
    return 1;
  }
  if (hipGetLastError() != hipSuccess) {
    cout << "CUDA Kernel Failed" << endl;
    return 1;
  }
  // Same as cuda_transformed_data
  float* int_transformed_data = (float*)malloc(n * sizeof(float));
  if (int_transformed_data == NULL) {
    cerr << "Memory Allocation Failed: int_transformed_data" << endl;
    return 1;
  }
  memset(int_transformed_data, 0, n * sizeof(int));
  if (hipMemcpy(int_transformed_data,
                 cuda_transformed_data,
                 n * sizeof(float),
                 hipMemcpyDeviceToHost) != hipSuccess) {
    cerr << "hipMemcpy() Failed" << endl;
    return 1;
  }
  // temp get transformed data
  for (int i = 0; i < n; i++) *(transformed_data + i) = (uint8_t)*(int_transformed_data + i);
  free(int_transformed_data);
  hipUnbindTexture(tex_ref);
  hipFreeArray(cuArray);
  if (hipFree(cuda_transformed_data) != hipSuccess) {
    cerr << "hipFree() Failed" << endl;
    return 1;
  }
  return 0;
}

int CUDAReformWrapper() {return 0;}
